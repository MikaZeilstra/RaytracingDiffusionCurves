#include "hip/hip_runtime.h"
#pragma once
#include<hip/hip_runtime_api.h>
#include<>
#include<stdio.h>

extern "C" __global__ void setFloatKernel(float* dest, unsigned int n, float src) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x){
        dest[i] = src;
    }
}

extern "C" __host__ void setFloatDevice(float* dest, unsigned int n, float src) {
   setFloatKernel << <256, 16 >> > (dest, n, src);
}

extern "C" __host__ void gaussianBlur(float4 * dest, float4 * source, float* strengths, int width, int height) {
}