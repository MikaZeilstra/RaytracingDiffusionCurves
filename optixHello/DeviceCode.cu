#include "hip/hip_runtime.h"
//
// Copyright (c) 2021, NVIDIA CORPORATION. All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NVIDIA CORPORATION nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//

#pragma once
#include <optix.h>
#include "params.h"
#include <hip/hip_runtime.h>

#define DU 1e-3

extern "C" {
__constant__ Params params;
}


//Returns the normal to the right of the curve direction
extern "C" __forceinline__ __device__ void calculateSplineNormal(float t, float3 * v, float2 & result) {
    result.x = (1 / 6.0f) * (3 * t * t * v[3].y + v[0].y * (-3 * t * t + 6 * t - 3) + v[1].y * (9 * t * t - 12 * t) + v[2].y * (-9 * t * t + 6 * t + 3));
    result.y = -(1 / 6.0f) * (3 * t * t * v[3].x + v[0].x * (-3 * t * t + 6 * t - 3) + v[1].x * (9 * t * t - 12 * t) + v[2].x * (-9 * t * t + 6 * t + 3));
}

//Returns true if ray is hitting the right side of curve
extern "C" __forceinline__ __device__ bool checkLeftRight(float t, float3 & ray_direction, float3 * v) {
    float2 curve_normal = {};
    calculateSplineNormal(t, v, curve_normal);

    return ((curve_normal.x * ray_direction.x + curve_normal.y * ray_direction.y) > 0);
}

extern "C" __global__ void __raygen__rg()
{
    const uint3 idx = optixGetLaunchIndex();
    unsigned int p0, p1, p2, p3;
    float4 color = {0,0,0,1};
    float3 ray_origin = {};
    float3 ray_direction = {};
    float weight_total = 0;
    float rot_cos, rot_sin;

    sincospif(2 / params.number_of_rays_per_pixel, &rot_sin, &rot_cos);


    ray_origin.x = idx.x * params.zoom_factor + params.offset_x;
    ray_origin.y = idx.y * params.zoom_factor + params.offset_y;
    ray_origin.z = 0;
    
    ray_direction.x = 1;
    ray_direction.y = 0;
    ray_direction.z = 0;  


    for (int i = 0; i < params.number_of_rays_per_pixel; i++) {

        
        
        

        optixTrace(
            params.traversable,
            ray_origin,
            ray_direction,
            0.0f,
            1e16f,
            0.0f,
            OptixVisibilityMask(255),
            OPTIX_RAY_FLAG_NONE,
            0,
            1,
            0,
            p0,p1,p2,p3
        );
        
        float4 result = { int_as_float(p0),int_as_float(p1), int_as_float(p2), int_as_float(p3) };

        //keep track of color
        weight_total += result.w;

        //Accumulate color
        color.x += result.x * result.w;
        color.y += result.y * result.w;
        color.z += result.z * result.w;


        if (idx.x == 150 && idx.y == 50) {
            //printf("rayO : %f, %f, %f \n", ray_origin.x, ray_origin.y, ray_origin.z);
            //printf("result : %f, %f, %f, %f \n", result.x, result.y, result.z, result.w);
        }
        //Rotate Ray
        ray_direction = { ray_direction.x * rot_cos - ray_direction.y * rot_sin,
                        ray_direction.x * rot_sin + ray_direction.y * rot_cos,
                        0 };


        
    }

    
    
    //Save average color
    params.image[idx.y * params.image_width + idx.x].x = color.x / weight_total;
    params.image[idx.y * params.image_width + idx.x].y = color.y / weight_total;
    params.image[idx.y * params.image_width + idx.x].z = color.z / weight_total;

}

extern "C" __global__ void __miss__ms() 
{
    const uint3 idx = optixGetLaunchIndex();
    /*
	optixSetPayload_0(float_as_int((idx.x % 256) / 256.0f));
	optixSetPayload_1(float_as_int((idx.y % 256) / 256.0f));
	optixSetPayload_2(float_as_int(((idx.x + idx.y + params.frame)  % 256) /256.0f));

    */

    optixSetPayload_0(float_as_int(0));
    optixSetPayload_1(float_as_int(0));
    optixSetPayload_2(float_as_int(0));
    optixSetPayload_3(float_as_int(0));
}

extern "C" __global__ void __closesthit__ch()
{
    float u = optixGetCurveParameter();
    float rt = optixGetRayTmax();
    int vertex_index = optixGetPrimitiveIndex();
    float3 ray_direction = optixGetWorldRayDirection();
    float3 ray_origin = optixGetWorldRayOrigin();


    float weight = 1;    


    
    if (checkLeftRight(u, ray_direction, &(params.vertices[params.segmentIndices[ vertex_index]]))) {
        optixSetPayload_0(float_as_int(0));
        optixSetPayload_1(float_as_int(0));
        optixSetPayload_2(float_as_int(0));
        optixSetPayload_3(float_as_int(weight));
    }
    else {
        optixSetPayload_0(float_as_int(1));
        optixSetPayload_1(float_as_int(1));
        optixSetPayload_2(float_as_int(1));
        optixSetPayload_3(float_as_int(weight));        
    }
}